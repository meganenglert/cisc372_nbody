#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include <hip/hip_runtime.h>

// compute: Updates the positions and locations of the objects in the system based on gravity.
// Parameters: None
// Returns: None
// Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
//__global__ void compute_help()
//{
//}
__global__ void initialize(int NUMENTITIES, vector3 *values, vector3 **accels)
{
	int index = threadIdx.x;
	int stride = blockDim.x;

	for (int i = index; i < NUMENTITIES; i += stride)
	{
		accels[i] = &values[i * NUMENTITIES];
	}
}

__global__ void fill(int NUMENTITIES, vector3 *values, vector3 **accels)
{
	for (int i = index; i < NUMENTITIES; i += stride)
	{
		// first compute the pairwise accelerations.  Effect is on the first argument.
		for (int j = 0; j < NUMENTITIES; j++)
		{
			if (i == j)
			{
				FILL_VECTOR(accels[i][j], 0, 0, 0);
			}
			else
			{
				vector3 distance;
				for (k = 0; k < 3; k++)
					distance[k] = hPos[i][k] - hPos[j][k];
				double magnitude_sq = distance[0] * distance[0] + distance[1] * distance[1] + distance[2] * distance[2];
				double magnitude = sqrt(magnitude_sq);
				double accelmag = -1 * GRAV_CONSTANT * mass[j] / magnitude_sq;
				FILL_VECTOR(accels[i][j], accelmag * distance[0] / magnitude, accelmag * distance[1] / magnitude, accelmag * distance[2] / magnitude);
			}
		}
	}
}
void compute()
{
	// make an acceleration matrix which is NUMENTITIES squared in size;
	int i, j, k;
	vector3 *values = (vector3 *)malloc(sizeof(vector3) * NUMENTITIES * NUMENTITIES);
	vector3 **accels = (vector3 **)malloc(sizeof(vector3 *) * NUMENTITIES);
	vector3 *d_values;
	hipMalloc((vector3 **)&d_values, sizeof(values));
	vector3 **d_accels;
	hipMalloc((vector3 ***)&d_accels, sizeof(accels));

	int numBlocks = (NUMENTITIES + blockSize - 1) / blockSize;
	construct_row<<<1, 256>>>(NUMENTITIES, d_values, d_accels);
	hipDeviceSynchronize();

	// sum up the rows of our matrix to get effect on each entity, then update velocity and position.
	for (i = 0; i < NUMENTITIES; i++)
	{
		vector3 accel_sum = {0, 0, 0};
		for (j = 0; j < NUMENTITIES; j++)
		{
			for (k = 0; k < 3; k++)
				accel_sum[k] += accels[i][j][k];
		}
		// compute the new velocity based on the acceleration and time interval
		// compute the new position based on the velocity and time interval
		for (k = 0; k < 3; k++)
		{
			hVel[i][k] += accel_sum[k] * INTERVAL;
			hPos[i][k] = hVel[i][k] * INTERVAL;
		}
	}
	free(accels);
	free(values);
	hipFree(d_accels);
	hipFree(d_values);
}
